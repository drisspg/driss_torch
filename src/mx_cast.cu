#include "hip/hip_runtime.h"
#include "cute/pointer.hpp"
#include "include/mx_cast.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/Float8_e4m3fn.h>
#include <hip/hip_cooperative_groups.h>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// CUTLASS includes
#include <cute/tensor.hpp>
#include <cutlass/util/device_memory.h>

#include <cmath>

namespace driss_torch_kernels {

using namespace cute;

template <typename inpt_type>
__device__ __forceinline__ __hip_fp8_storage_t
convert_to_fp8(inpt_type scaled_input, __hip_fp8_interpretation_t fp8_dtype) {
  return __hip_cvt_float_to_fp8(scaled_input, __hip_saturation_t::__HIP_SATFINITE,
                               fp8_dtype);
}

__device__ __forceinline__ float compute_row_scales(const float abs_val, int row_idx, int col_idx, auto num_rows) {
  typedef hipcub::WarpReduce<int> WarpReduce;  // Changed to int reduction
  __shared__ typename WarpReduce::TempStorage temp_storage[num_rows];

  // Compute log2 and ceiling to next integer immediately
  float log2_val_float = log2f(abs_val);
  int log2_val = (int)ceilf(log2_val_float);  // Round up to next integer

  // Perform reduction using integers
  int max_log2 = WarpReduce(temp_storage[row_idx]).Reduce(log2_val, hipcub::Max());
  max_log2 = __shfl_sync(0xffffffff, max_log2, 0);

  // For E4M3, subtract log2(256.0) = 8 as integer
  int scale_log2 = max_log2 - 8;

  // Convert back to float only at the end
  return powf(2.0f, (float)scale_log2);
}


template <class Element, class TensorInput, class TensorOutput,
          class TensorScale, class ThrShape, class BlockShape>
__global__ void mx_fp8_quantize_kernel(TensorInput input, TensorOutput output,
                                       TensorScale scale, ThrShape thr_shape, BlockShape blck_shape) {

  auto tid = threadIdx.x + threadIdx.y * blockDim.x;
  // Slice the tensors to obtain a view into each tile.
  Tensor tile_input = input(make_coord(_, _), blockIdx.x, blockIdx.y);
  Tensor tile_output = output(make_coord(_, _), blockIdx.x, blockIdx.y);
  Tensor tile_scale = scale(make_coord(_, _), blockIdx.x, blockIdx.y);

  constexpr auto num_rows = size<0>(thr_shape);
  const int num_tiles = get<0>(blck_shape)/num_rows;

  // Tile divide for input
  auto tiled_input_tensor = tiled_divide(tile_input, thr_shape);
  auto tiled_ouput_tenosr = tiled_divide(tile_output, thr_shape);
  auto tiled_scale_tensor_e8m0 = tiled_divide(tile_scale, make_shape(get<0>(thr_shape), 1));

  auto row_idx = threadIdx.y;
  auto col_idx = threadIdx.x;

  #pragma unroll
  for (auto i = 0; i < num_tiles; i++) {

    auto sub_tiled_input = tiled_input_tensor(make_coord(_, _), i, 0);
    auto sub_tile_out = tiled_ouput_tenosr(make_coord(_, _), i, 0);
    auto sub_tiled_scale_e8m0 = tiled_scale_tensor_e8m0(make_coord(_, _), i, 0);

    auto inpt = sub_tiled_input(row_idx, col_idx);

    // Calculate absolute values
    auto abs = std::abs(static_cast<float>(inpt));

    // // Compute row scales
    auto scale = compute_row_scales(abs, row_idx, col_idx, num_rows);
    __syncthreads();

    // // Apply scaling and convert to FP8
    auto inverse_scale = 1 / scale;
    auto scaled = static_cast<float>(inpt) * inverse_scale;
    auto out = convert_to_fp8(scaled, __HIP_E4M3_FNUZ);

    // Write output directly to global memory
    sub_tile_out(row_idx, col_idx) = out;

    // Write scale directly to global memory (only one thread per row)
    if (col_idx == 0) {
      auto converted = __nv_cvt_float_to_e8m0(scale, __HIP_SATFINITE, hipRoundMode::hipRoundPosInf);
      sub_tiled_scale_e8m0(row_idx, 0) = converted;
    }
  }

}

} // namespace driss_torch_kernels

namespace driss_torch {
using namespace cute;

std::tuple<at::Tensor, at::Tensor> mx_fp8_quantize(at::Tensor input,
                                                   int64_t block_size,
                                                   int64_t axis, bool transpose,
                                                   c10::ScalarType fp8_type) {

  TORCH_CHECK(input.is_cuda(), "Input tensor must be a CUDA tensor");
  TORCH_CHECK(input.is_contiguous(), "Input tensor must be contiguous");
  TORCH_CHECK(input.scalar_type() == at::kHalf ||
                  input.scalar_type() == at::kFloat ||
                  input.scalar_type() == at::kBFloat16,
              "Input tensor must be float, half, or bfloat16");
  TORCH_CHECK(block_size > 0 && block_size <= 32,
              "Block size must be positive and <= 32");
  TORCH_CHECK(input.dim() == 2,
              "Input tensor must be 2D for CUTLASS implementation");
  TORCH_CHECK(fp8_type == at::kFloat8_e4m3fn || fp8_type == at::kFloat8_e5m2,
              "FP8 type must be Float8_e4m3fn or Float8_e5m2");

  // Get tensor dimensions
  auto input_shape = input.sizes();
  auto input_strides = input.strides();

  TORCH_CHECK(input.dim() == 2, "Only support 2d tensor for now");
  int64_t m = input_shape[0];
  int64_t n = input_shape[1];
  auto num_k_blocks = cutlass::ceil_div(n, 32);
  auto total_blocks = m * num_k_blocks;

  // Create output tensors
  auto output = at::empty_like(input, input.options().dtype(fp8_type));
  auto scale =
      at::empty({total_blocks}, input.options().dtype(at::kFloat8_e8m0fnu));

  auto tensor_input_shape = make_shape(m, n);
  auto tensor_scale_shape = make_shape(m, num_k_blocks);

  auto input_ptr = static_cast<__hip_bfloat16 *>(input.data_ptr());
  auto scale_ptr = static_cast<__hip_fp8_storage_t *>(scale.data_ptr());
  auto output_ptr = static_cast<__hip_fp8_storage_t *>(output.data_ptr());

  Tensor tensor_input =
      make_tensor(make_gmem_ptr(input_ptr), make_layout(tensor_input_shape, LayoutRight()));
  Tensor tensor_scale =
      make_tensor(make_gmem_ptr(scale_ptr), make_layout(tensor_scale_shape, LayoutRight()));
  Tensor tensor_ouput =
      make_tensor(make_gmem_ptr(output_ptr), make_layout(tensor_input_shape, LayoutRight()));

  // Keep it easy for now
  auto block_shape = make_shape(Int<128>{}, Int<32>{});
  auto scale_shape = make_shape(get<0>(block_shape), Int<1>{});
  TORCH_CHECK(evenly_divides(tensor_input_shape, block_shape),
              "Need block shape to evenly divide the input tensor for now");


  // These will be used to determine the CUDA kernel grid dimensions.
  Tensor tiled_tensor_input = tiled_divide(tensor_input, block_shape);
  Tensor tiled_tensor_ouput = tiled_divide(tensor_ouput, block_shape);
  Tensor tiled_tensor_scale = tiled_divide(tensor_scale, scale_shape);

  // Thread arrangement
  auto thr_shape = make_shape(Int<8>{}, Int<32>{});
  dim3 gridDim(
      size<1>(tiled_tensor_input),
      size<2>(tiled_tensor_input));
  dim3 blockDim(size<1>(thr_shape), size<0>(thr_shape));

  //
  // Launch the kernel
  //
  driss_torch_kernels::mx_fp8_quantize_kernel<__hip_bfloat16>
      <<<gridDim, blockDim>>>(tiled_tensor_input, tiled_tensor_ouput,
                              tiled_tensor_scale, thr_shape, block_shape);

  // Check for CUDA errors
  C10_CUDA_CHECK(hipGetLastError());

  return {output, scale};
}

} // namespace driss_torch
